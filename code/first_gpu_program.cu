// first_gpu_program.cu

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 65000

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

__global__ void add(int *a, int *b, int *c)
{
	int tid = blockIdx.x;
	if (tid < N)
		c[tid] = a[tid] + b[tid];
}

int main(int argc, char **argv)
{
	int a[N], b[N], c[N];
	int *dev_a, *dev_b, *dev_c;

	// Allocate memory on the GPU
	HANDLE_ERROR(hipMalloc((void**) &dev_a, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_b, N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**) &dev_c, N * sizeof(int)));

	// Fill the arrays 'a' and ''b' on the CPU
	for (int i = 0; i < N; i++) {
		a[i] = -i;
		b[i] = i * i;
	}

	// Copy the arrays 'a' and 'b' to the GPU
	HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int),
					hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int),
					hipMemcpyHostToDevice));

	add<<<N, 1>>>(dev_a, dev_b, dev_c);

	// Copy array 'c' back from the GPU to the CPU
	HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int),
					hipMemcpyDeviceToHost));

	// Display the results
	for (int i = 0; i < N; i++) {
		if (i % 256 == 0)
			printf("%d + %d = %d\n", a[i], b[i], c[i]);
	}

	// Free the memory allocated on the GPU
	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	return 0;
}

